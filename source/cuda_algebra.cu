#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <stdio.h>
#include <hipsolver.h>

extern "C" {
#include "cuda_algebra.h"
}

extern "C"
double cuda_norm(double *d_A, int m, int n, int lda)
{
    hipsolverHandle_t cusolverH = NULL;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;

    // Allocate S array where to retrieve singular values.
    double *S = (double *) malloc(sizeof(double) * n);

    double *d_S = NULL;
    int *devInfo = NULL;
    double *d_work = NULL;
    double *d_rwork = NULL;

    int lwork = 0;

    // Create cusolverDn handle.
    cusolver_status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    // Create a copy of given matrix, since it is going to be destroyed after
    // calculating SVD.
    double *d_A_cp = NULL;
    cudaStat1 = hipMalloc((void **) &d_A_cp, sizeof(double)*lda*n);
    assert(hipSuccess == cudaStat1);
    cudaStat1 =
        hipMemcpy(d_A_cp, d_A, sizeof(double)*lda*n, hipMemcpyDeviceToDevice);
    assert(hipSuccess == cudaStat1);
    d_A = d_A_cp;

    // Allocate helper matrixes and utils.
    cudaStat1 = hipMalloc((void **) &d_S, sizeof(double)*n);
    cudaStat2 = hipMalloc((void **) &devInfo, sizeof(int));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

    // Query working space of SVD.
    cusolver_status = hipsolverDnDgesvd_bufferSize(
            cusolverH,
            m,
            n,
            &lwork);
    assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    cudaStat1 = hipMalloc((void **) &d_work, sizeof(double)*lwork);
    assert(hipSuccess == cudaStat1);

    // Compute SVD.
    signed char jobu = 'N';  // all m columns of U
    signed char jobvt = 'N';  // all n columns of VT
    cusolver_status = hipsolverDnDgesvd(
            cusolverH,
            jobu,
            jobvt,
            m,
            n,
            d_A,
            lda,
            d_S,
            NULL,
            lda,  // ldu
            NULL,
            lda,  // ldvt
            d_work,
            lwork,
            d_rwork,
            devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(S, d_S, sizeof(double)*n, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    // Calculate frobenius norm.
    double frob = 0.0;
    for (int i = 0; i < n; i++) frob += powf(S[i], 2.0);
    frob = sqrt(frob);

    if (cusolverH) hipsolverDnDestroy(cusolverH);

    // Release resources.
    free(S);
    if (d_S) hipFree(d_S);
    if (devInfo) hipFree(devInfo);
    if (d_work) hipFree(d_work);
    if (d_rwork) hipFree(d_rwork);
    if (d_A_cp) hipFree(d_A_cp);

    return frob;
}
